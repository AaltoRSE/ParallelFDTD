#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
//
// This file is a part of the PadallelFDTD Finite-Difference Time-Domain
// simulation library. It is released under the MIT License. You should have 
// received a copy of the MIT License along with ParallelFDTD.  If not, see
// http://www.opensource.org/licenses/mit-license.php
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//
// For details, see the LICENSE file
//
// (C) 2013-2014 Jukka Saarelma
// Aalto University School of Science
//
///////////////////////////////////////////////////////////////////////////////

#include "cudaUtils.h"
#include "cudaMesh.h"

void CudaMesh::setupMesh(unsigned char* d_position_ptr,
                         unsigned char* d_material_ptr,
                         unsigned int number_of_unique_materials,
                         float* material_coefficients,
                         float* parameter_ptr,
                         uint3 voxelization_dim,
                         uint3 block_size, 
                         unsigned int element_type) {
  
  clock_t start_t;
  clock_t end_t;
  start_t = clock();                            

  // Pad the mesh to mach the block size of choice
  padWithZeros(&d_position_ptr, 
               &d_material_ptr, 
               &voxelization_dim, 
               block_size.x, 
               block_size.y, 
               block_size.z);

  this->num_elements_ = voxelization_dim.x*voxelization_dim.y*voxelization_dim.z;
  this->dim_x_ = voxelization_dim.x;
  this->dim_y_ = voxelization_dim.y;
  this->dim_z_ = voxelization_dim.z;
  this->dim_xy_ = voxelization_dim.x*voxelization_dim.y;
  this->block_size_x_ = block_size.x;
  this->block_size_y_ = block_size.y;
  this->block_size_z_ = block_size.z;
  
  this->h_material_coef_ptr_ = material_coefficients;
  this->h_parameter_ptr_ = parameter_ptr;
  
  this->number_of_unique_materials_ = number_of_unique_materials;

  this->position_idx_ptr_.push_back(d_position_ptr);
  this->material_idx_ptr_.push_back(d_material_ptr);

  c_log_msg(LOG_INFO, 
            "CudaMesh::setupCudaMesh:- dim x: %d y: %d z: %d num elements %d",
            voxelization_dim.x, voxelization_dim.y, voxelization_dim.z, this->num_elements_);

  //////////// Translate the node data to given boundary formulation
  start_t = clock();
  if(element_type == 0 || element_type == 1 || element_type == 3)
    this->toBilbaoScheme( );
  else  
    this->toKowalczykScheme();

  end_t = clock()-start_t;
  c_log_msg(LOG_INFO,"CudaMesh::Voxelization nodes to scheme done  - time: %f seconds",
        ((float)end_t/CLOCKS_PER_SEC));

  c_log_msg(LOG_INFO, "CudaMesh::Voxelization done");
  printMemInfo("voxelizeGeometryDevice memory before return", getCurrentDevice());
    
  cudasafe(hipPeekAtLastError(),
           "CudaMesh::voxelizeGeometryToDevice:" 
           "voxelizerDevice - peek before return");
  
  cudasafe(hipDeviceSynchronize(), 
           "CudaMesh::voxelizeGeometryToDevice:" 
           "voxelizerDevice - hipDeviceSynchronize at before return");  
}


void CudaMesh::setupMeshDouble(unsigned char* d_position_ptr,
                               unsigned char* d_material_ptr,
                               unsigned int number_of_unique_materials,
                               double* material_coefficients,
                               double* parameter_ptr,
                               uint3 voxelization_dim,
                               uint3 block_size, 
                               unsigned int element_type) {
  
  clock_t start_t;
  clock_t end_t;
  start_t = clock();                            

  // Pad the mesh to match the block size of choice
  padWithZeros(&d_position_ptr, 
               &d_material_ptr, 
               &voxelization_dim, 
               block_size.x, 
               block_size.x, 
               block_size.z);

  this->num_elements_ = voxelization_dim.x*voxelization_dim.y*voxelization_dim.z;
  this->dim_x_ = voxelization_dim.x;
  this->dim_y_ = voxelization_dim.y;
  this->dim_z_ = voxelization_dim.z;
  this->dim_xy_ = voxelization_dim.x*voxelization_dim.y;
  this->block_size_x_ = block_size.x;
  this->block_size_y_ = block_size.y;
  this->block_size_z_ = block_size.z;
  
  this->h_material_coef_ptr_double_ = material_coefficients;
  this->h_parameter_ptr_double_ = parameter_ptr;
  this->number_of_unique_materials_ = number_of_unique_materials;
  this->position_idx_ptr_.push_back(d_position_ptr);
  this->material_idx_ptr_.push_back(d_material_ptr);

  c_log_msg(LOG_INFO, 
            "CudaMesh::setupCudaMesh:- dim x: %d y: %d z: %d num elements %d",
            voxelization_dim.x, voxelization_dim.y, voxelization_dim.z, this->num_elements_);

  //////////// Translate the node data to given boundary formulation
  start_t = clock();
  if(element_type == 0 || element_type == 1)
    this->toBilbaoScheme();
  else  
    this->toKowalczykScheme();

  end_t = clock()-start_t;
  c_log_msg(LOG_INFO,"CudaMesh::Voxelization nodes to scheme done  - time: %f seconds",
        ((float)end_t/CLOCKS_PER_SEC));

  c_log_msg(LOG_INFO, "CudaMesh::Voxelization done");
  printMemInfo("voxelizeGeometryDevice memory before return", getCurrentDevice());
    
  cudasafe(hipPeekAtLastError(),
           "CudaMesh::voxelizeGeometryToDevice:" 
           "voxelizerDevice - peek before return");
  
  cudasafe(hipDeviceSynchronize(), 
           "CudaMesh::voxelizeGeometryToDevice:" 
           "voxelizerDevice - hipDeviceSynchronize at before return");  
}

void CudaMesh::toKowalczykScheme() {
  c_log_msg(LOG_INFO, "CudaMesh::toKowalczykScheme -  translate to kowalczyk");
  
  unsigned int num_elems = this->getNumberOfElements();
  int threadsPerBlock = 512;
  dim3 block_dim(threadsPerBlock);

  int numBlocks = (num_elems + threadsPerBlock - 1) / threadsPerBlock;
  dim3 grid_dim((unsigned int)ceil(sqrt(numBlocks)), (unsigned int)ceil(sqrt(numBlocks))); 

  c_log_msg(LOG_VERBOSE, "CudaMesh::nodes2KowalczykScheme - Grid x: %u y: %u z: %u", 
                          grid_dim.x, grid_dim.y, grid_dim.z);
  

  toKowalczykKernel<<<grid_dim, block_dim>>>(this->position_idx_ptr_.at(0), 
                                             this->material_idx_ptr_.at(0), 
                                             num_elems);

  // Calculate number of boundaries
  unsigned int* h_air_elements = NULL;
  unsigned int* h_boundary_elements = NULL;

  unsigned int* d_air_nodes = valueToDevice(1, (unsigned int)0, 0);
  unsigned int* d_boundary_nodes = valueToDevice(1, (unsigned int)0, 0);

  calcBoundaries<<<grid_dim, block_dim>>>(this->position_idx_ptr_.at(0), 
                                          d_air_nodes, d_boundary_nodes, 0x80, 0,
                                          num_elems);
  
  h_air_elements = fromDevice(1, d_air_nodes,0);
  h_boundary_elements= fromDevice(1, d_boundary_nodes,0);
  
  destroyMem(d_air_nodes);
  destroyMem(d_boundary_nodes);

  // Assign counted values to the class
  this->num_air_elements_total_ = *h_air_elements;
  this->num_boundary_elements_total_ = *h_boundary_elements;
  free(h_air_elements);
  free(h_boundary_elements);

  c_log_msg(LOG_DEBUG, "CudaMesh::toKowalczykScheme - air elements: %u boundary elements: %u",
                        this->num_air_elements_total_, 
                        this->num_boundary_elements_total_);

  cudasafe(hipDeviceSynchronize(), 
           "CudaMesh::toKowalczykScheme - hipDeviceSynchronize at before return");
}

void CudaMesh::toBilbaoScheme() {
  c_log_msg(LOG_INFO, "CudaMesh::nodes2BilbaoScheme -  voxelizerDevice translate to bilbao");

  unsigned int num_elems = this->getNumberOfElements();
  
  int threadsPerBlock = 512;
  dim3 block_dim(threadsPerBlock);

  int numBlocks = (num_elems + threadsPerBlock - 1) / threadsPerBlock;
  dim3 grid_dim((unsigned int)ceil(sqrt(numBlocks)), (unsigned int)ceil(sqrt(numBlocks))); 

  c_log_msg(LOG_INFO, "CudaMesh::nodes2Bilbao - Grid x: %u y: %u z: %u", 
                         grid_dim.x, grid_dim.y, grid_dim.z);
  
  toBilbaoKernel<<<grid_dim, block_dim>>>(this->position_idx_ptr_.at(0), 
                                          this->material_idx_ptr_.at(0), 
                                          num_elems);
  unsigned int* h_air_elements = NULL;
  unsigned int* h_boundary_elements = NULL;

  // Calculate number of boundaries
  unsigned int* d_air_nodes = valueToDevice(1, (unsigned int)0, 0);
  unsigned int* d_boundary_nodes = valueToDevice(1, (unsigned int)0, 0);

  calcBoundaries<<<grid_dim, block_dim>>>(this->position_idx_ptr_.at(0), 
                                          d_air_nodes, d_boundary_nodes, 0x86, 0,
                                          num_elems);
   
  h_air_elements = fromDevice(1, d_air_nodes,0);
  h_boundary_elements = fromDevice(1, d_boundary_nodes,0);
  c_log_msg(LOG_INFO, "CudaMesh::nodes2Bilbao - returned air elems: %u, boundary: %u", *h_air_elements, *h_boundary_elements);

  destroyMem(d_air_nodes);
  destroyMem(d_boundary_nodes);
  
  this->num_air_elements_total_ = *h_air_elements;
  this->num_boundary_elements_total_ = *h_boundary_elements;
  free(h_air_elements);
  free(h_boundary_elements);

  c_log_msg(LOG_DEBUG, "CudaMesh::nodes2BilbaoScheme - air elements: %u boundary elements: %u",
        this->num_air_elements_total_, this->num_boundary_elements_total_);

  cudasafe(hipDeviceSynchronize(), 
          "CudaMesh<LongNode>::nodes2BilbaoScheme - hipDeviceSynchronize at before return");

  printCheckSum(this->getPositionIdxPtrAt(0), this->num_elements_, "node2Bilbao - return");
}

void padWithZeros(unsigned char** d_mesh, uint3* dim, 
                  unsigned int block_size_x, 
                  unsigned int block_size_y,
                  unsigned int block_size_z) {

  unsigned int padX = 0;
  unsigned int padY = 0;
  unsigned int padZ = 0;

  unsigned int dim_x = (*dim).x;
  unsigned int dim_y = (*dim).y;
  unsigned int dim_z = (*dim).z;

  // Pad so that the dimensions are even with the grid size
  if((dim_x+padX)%block_size_x != 0)
    padX += (block_size_x-((dim_x+padX)%block_size_x));

  if((dim_y+padY)%block_size_y != 0)
    padY += (block_size_y-((dim_y+padY)%block_size_y));

  if((dim_z+padZ)%block_size_z != 0)
    padZ += (block_size_z-((dim_z+padZ)%block_size_z));

  // New size with the padding
  int newSize = (dim_x+padX)*(dim_y+padY)*(dim_z+padZ);

  unsigned char* d_mesh_new = valueToDevice(newSize, (unsigned char)0, 0);
  
  dim3 block(block_size_x, block_size_y, block_size_z);
  dim3 grid((int)ceil((float)(dim_x+padX)/(float)block.x), 
            (int)ceil((float)(dim_y+padY)/(float)block.y), 
          dim_z);

  padWithZerosKernel<<<grid, block>>>(d_mesh_new, *d_mesh, 
                                      dim_x, dim_y, dim_z,
                                      padX, padY, padZ, 0);

  destroyMem(*d_mesh);
  (*d_mesh) = d_mesh_new;

  dim_x = dim_x+padX;
  dim_y = dim_y+padY;
  dim_z = dim_z+padZ;

  // update dimensions
  uint3 dim_;
  dim_.x = dim_x;
  dim_.y = dim_y;
  dim_.z = dim_z;
  cudasafe(hipDeviceSynchronize(), "cudaUtils.cu: padWithZeros -hipDeviceSynchronize at before return");
  *dim = dim_;
}

void padWithZeros(unsigned char** d_position_ptr, 
                  unsigned char**d_material_ptr, 
                  uint3* dim, 
                  unsigned int block_size_x, 
                  unsigned int block_size_y, 
                  unsigned int block_size_z) {

  c_log_msg(LOG_INFO, "cudaUtils.cu: padWithZeros - Begin");
  uint3 dim_ = *dim;
  
  padWithZeros(d_position_ptr, &dim_, block_size_x, block_size_y, block_size_z);

  dim_ = *dim;

  padWithZeros(d_material_ptr, &dim_, block_size_x, block_size_y, block_size_z);

  c_log_msg(LOG_INFO, "cudaUtils.cu: padWithZeros - Return");
  // update dimensions
  *dim = dim_;

}

__host__ __device__ void toBilbao(unsigned char* d_position_ptr, 
                                  unsigned char* d_material_ptr) {
  unsigned int k = (unsigned int)*d_position_ptr;

    if(k == 0){
      *d_position_ptr = (unsigned char)0;
      *d_material_ptr = (unsigned char)0;
      return;
    }

    if(k <= 8) {
      *d_position_ptr = (unsigned char)3;
      *d_position_ptr |= 0x80;
      return;
    }

    if(k > 8 && k <= 20) {
      *d_position_ptr = (unsigned char)4;
      *d_position_ptr |= 0x80;
      return;
    }

    if(k > 20 && k <= 26) {
      *d_position_ptr = (unsigned char)5;
      *d_position_ptr |= 0x80;
      return;
    }
    
    if(k ==27) {
      *d_position_ptr = (unsigned char)6;
      *d_position_ptr |= 0x80;
      return;
    }
}

__host__ __device__ void toKowalczyk(unsigned char* d_position_ptr, 
                                     unsigned char* d_material_ptr) {
  unsigned int k = (unsigned int)*d_position_ptr;
  
  if(k == 0){
    *d_position_ptr = (unsigned char)0;
    *d_material_ptr = (unsigned char)0;
    return;
  }
  if(k == 1) { // down left in
    *d_position_ptr = 0|SIGN_Z|DIR_X|DIR_Y|DIR_Z|CENTERED_MASK;
    return;
  }
  if(k == 2) { // down right in
    *d_position_ptr = 0|SIGN_Z|SIGN_X|DIR_X|DIR_Y|DIR_Z|CENTERED_MASK;
    return;
  }
  if(k == 3) { // down left out
    *d_position_ptr = 0|SIGN_Z|SIGN_Y|DIR_X|DIR_Y|DIR_Z|CENTERED_MASK;
    return;
  }
  if(k == 4) { // down right out
    *d_position_ptr = 0|SIGN_Z|SIGN_Y|SIGN_X|DIR_X|DIR_Y|DIR_Z|CENTERED_MASK;
    return;
  }
  if(k == 5) { //  up left in
    *d_position_ptr = 0|DIR_X|DIR_Y|DIR_Z|CENTERED_MASK;
    return;
  }
  if(k == 6) { //  up right in
    *d_position_ptr = 0|SIGN_X|DIR_X|DIR_Y|DIR_Z|CENTERED_MASK;
    return;
  }
  if(k == 7) { // up left out
    *d_position_ptr = 0|SIGN_Y|DIR_X|DIR_Y|DIR_Z|CENTERED_MASK;
    return;
  }
  if(k == 8) { // up right out
    *d_position_ptr = 0|SIGN_Y|SIGN_X|DIR_X|DIR_Y|DIR_Z|CENTERED_MASK;
    return;
  }
  if(k == 9) { // 09 = Down, Left, Right, In 
    *d_position_ptr = 0|DIR_Y|DIR_Z|SIGN_Z|CENTERED_MASK;
    return;
  }
  if(k == 10) { // 10 = Down, Left, Right, Out
    *d_position_ptr = 0|DIR_Y|DIR_Z|SIGN_Z|SIGN_Y|CENTERED_MASK;
    return;
  }
  if(k == 11) { // 11 = Down, Left, In, Out 
    *d_position_ptr = 0|DIR_X|DIR_Z|SIGN_Z|CENTERED_MASK;
    return;
  }
  if(k == 12) { // 12 = Down, Right, In, Out 
    *d_position_ptr = 0|DIR_X|DIR_Z|SIGN_Z|SIGN_X|CENTERED_MASK;
    return;
  }
  if(k == 13){ // 13 = Up, Left, Right, In 
    *d_position_ptr = 0|DIR_Y|DIR_Z|CENTERED_MASK;
    return;
  }
  if(k == 14) { // 14 = Up, Left, Right, Out 
    *d_position_ptr = 0|DIR_Y|DIR_Z|SIGN_Y|CENTERED_MASK;
    return;
  }
  if(k == 15) { // 15 = Up, Left, In, Out 
    *d_position_ptr = 0|DIR_X|DIR_Z|CENTERED_MASK;
    return;
  }
  if(k == 16) { // 16 = Up, Right, In, Out 
    *d_position_ptr = 0|DIR_X|DIR_Z|SIGN_X|CENTERED_MASK;
    return;
  }
  if(k == 17) { // 17 = Up, Down, Left, In 
    *d_position_ptr = 0|DIR_Y|DIR_X|CENTERED_MASK;
    return;
  }
  if(k == 18) { // 18 = Up, Down, Right, In 
    *d_position_ptr = 0|DIR_X|DIR_Y|SIGN_X|CENTERED_MASK;
    return;
  }
  if(k == 19) { // 19 = Up, Down, Left, Out 
    *d_position_ptr = 0|DIR_X|DIR_Y|SIGN_Y|CENTERED_MASK;
    return;
  }
  if(k == 20) { // 20 = Up, Down, Right, Out 
    *d_position_ptr = 0|DIR_Y|DIR_X|SIGN_Y|SIGN_X|CENTERED_MASK;
    return;
  }
  if(k == 21) { // 21 = Left, Right, In, Out, Down 
    *d_position_ptr = 0|DIR_Z|SIGN_Z|CENTERED_MASK;
    return;
  }
  if(k == 22) { // 22 = Left, Right, Out, Down, Up 
    *d_position_ptr =  0|DIR_Y|SIGN_Y|CENTERED_MASK;
    return;
  }
  if(k == 23) { // 23 = Left, Right, In, Down, Up 
    *d_position_ptr = 0|DIR_Y|CENTERED_MASK;
    return;
  }
  if(k == 24) { // 24 = Right, In, Out, Down, Up 
    *d_position_ptr = 0|DIR_X|SIGN_X|CENTERED_MASK;
    return;
  }
  if(k == 25) { // 25 = Left, In, Out, Down, Up 
    *d_position_ptr = 0|DIR_X|CENTERED_MASK;
    return;
  }
  if(k == 26) { // 26 = Left, Right, In, Out, Up 
    *d_position_ptr = 0|DIR_Z|CENTERED_MASK;
    return;
  }
  if(k == 27) { // 27 = Left, Right, In, Out, Down, Up - AIR NODE
    *d_position_ptr = 0|CENTERED_MASK;
    return;
  }
}

__global__ void toKowalczykKernel(unsigned char* d_position_ptr, 
                                  unsigned char* d_material_ptr, 
                                  unsigned int num_elems) {
  int idx = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < num_elems) {
    toKowalczyk(d_position_ptr+idx, d_material_ptr+idx);
  }
}


__global__ void toBilbaoKernel(unsigned char* d_position_ptr, unsigned char* d_material_ptr, 
                               unsigned int num_elems) {
  int idx = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < num_elems) {
    toBilbao(d_position_ptr+idx, d_material_ptr+idx);
  }
}

__global__ void calcBoundaries(unsigned char* d_position_ptr, unsigned int* air, 
                               unsigned int* boundary, 
                               unsigned char air_value, 
                               unsigned char outside_value,
                               unsigned int num_elems) {
  unsigned int idx = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < num_elems) {
    if(d_position_ptr[idx] == air_value) {
      atomicAdd(air,1);
    }
    if(d_position_ptr[idx] != outside_value && d_position_ptr[idx] != air_value) {
      atomicAdd(boundary,1);
    }
  }
}

__global__ void padWithZerosKernel(unsigned char* d_mesh_new, 
                                   unsigned char* d_mesh_old,
                                   unsigned int dim_x, unsigned int dim_y, 
                                   unsigned int dim_z, 
                                   unsigned int block_x, unsigned int block_y, 
                                   unsigned int block_z, 
                                   unsigned int slice) {

  unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned int z = blockIdx.z;

  if(x < dim_x+1 && x > 0 && y < dim_y+1 &&  y > 0 && z < dim_z+1 && z > 0) {
    unsigned int old_idx = (z)*(dim_x)*(dim_y)+(y)*dim_x+x;
    unsigned int new_idx = z*(dim_x+block_x)*(dim_y+block_y)+y*(dim_x+block_x)+x;

    d_mesh_new[new_idx] = d_mesh_old[old_idx];
  }
}

